#include "hip/hip_runtime.h"
#include <cmath>
#include "core/smpl/def.h"
#include "core/smpl/smpl.h"
#include "common/common_types.h"

namespace smpl {
    namespace device {
        __global__ void FindKNN1(
                const PtrSz<const float> templateRestShape,
                const PtrSz<const float> shapeBlendShape,
                const int vertexnum,
                const PtrSz<const float> curvertices,
                PtrSz<float> dist
        ) {
            int i = blockIdx.x;
            int j = threadIdx.x;
            int ind = i * vertexnum + j;
            dist[ind] = 0;
            for (int k = 0; k < 3; k++) {
                float restShape = templateRestShape[j * 3 + k] + shapeBlendShape[j * 3 + k];
                dist[ind] += (curvertices[i * 3 + k] - restShape) * (curvertices[i * 3 + k] - restShape);
            }
        }

        __global__ void FindKNN2(
                const PtrSz<const float>dist,
                const int vertexnum,
                PtrSz<int> ind
        ) {
            int i = threadIdx.x;

            ind[i * 4 + 0] = 0;
            ind[i * 4 + 1] = 1;
            ind[i * 4 + 2] = 2;
            ind[i * 4 + 3] = 3;

            for (int l = 0; l < 4; l++)
                for (int p = 0; p < 3 - l; p++)
                    if (dist[i * vertexnum + ind[i * 4 + p]] > dist[i * vertexnum + ind[i * 4 + p + 1]]) {
                        int tmp = ind[p];
                        ind[p] = ind[p + 1];
                        ind[p + 1] = tmp;
                    }

            //find first 4 minimum distances
            for (int k = 4; k < vertexnum; k++)
                for (int t = 0; t < 4; t++)
                    if (dist[i * vertexnum + k] < dist[i * vertexnum + ind[i * 4 + t]]) {
                        for (int l = 3; l > t; l--)
                            ind[i * 4 + l] = ind[i * 4 + l - 1];
                        ind[i * 4 + t] = k;
                        continue;
                    }
        }

        __global__ void CalculateWeights(
                const PtrSz<const float> dist,
                const PtrSz<const float> weights,
                const PtrSz<const int> ind,
                const int jointnum,
                const int vertexnum,
                PtrSz<float>new_weights
        ) {
            int j = threadIdx.x; // num of weight
            int i = blockIdx.x; // num of vertex

            new_weights[i * jointnum + j] = 0;
            float weight = 0;
            for (int k = 0; k < 4; k++) {
                weight += dist[i * vertexnum + ind[i * 4 + k]];
                new_weights[i * jointnum + j] += dist[i * vertexnum + ind[i * 4 + k]] *
                        weights[ind[i * 4 + k] * jointnum + j];
            }
            new_weights[i * jointnum + j] /= weight;
        }
    }

    DeviceArray<float> SMPL::lbs_for_custom_vertices(
            const DeviceArray<float> &beta,
            const DeviceArray<float> &theta,
            const DeviceArray<float> &d_vertices
    ) {
        DeviceArray<float> d_shapeBlendShape(DeviceArray<float>(VERTEX_NUM * 3));
        DeviceArray<float> d_dist(DeviceArray<float>(d_vertices.size() * VERTEX_NUM));
        DeviceArray<int> d_ind(DeviceArray<int>(d_vertices.size() * 4));
        DeviceArray<float> d_cur_weights(DeviceArray<float>(d_vertices.size() * JOINT_NUM));
        DeviceArray<float> d_result_vertices(DeviceArray<float>(d_vertices.size() * 3));

        shapeBlendShape(beta, d_shapeBlendShape);
        // find k nearest neigbours
        device::FindKNN1<<<vertnum,VERTEX_NUM>>>(d_templateRestShape, d_shapeBlendShape, VERTEX_NUM, d_vertices, d_dist);
        device::FindKNN2<<<1,vertnum>>>(d_dist, VERTEX_NUM, d_ind);
        // calculate weights
        device::CalculateWeights<<<vertnum,JOINT_NUM>>>(d_dist, d_weights, d_ind,  JOINT_NUM, VERTEX_NUM, d_cur_weights);
        run(beta, theta, d_cur_weights, d_result_vertices, d_vertices);

        d_shapeBlendShape.release();
        d_dist.release();
        d_ind.release();
        d_cur_weights.release();

        return d_result_vertices;
    }
}