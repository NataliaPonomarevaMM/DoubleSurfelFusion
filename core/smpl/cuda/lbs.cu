#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include "core/smpl/def.h"
#include "core/smpl/smpl.h"
#include "common/common_types.h"
#include "common/Constants.h"

namespace surfelwarp {
    namespace device {
        __global__ void FindKNN1(
                const PtrSz<const float> templateRestShape,
                const PtrSz<const float> shapeBlendShape,
                const int vertexnum,
                const PtrSz<const float> curvertices,
                PtrSz<float> dist
        ) {
            int i = blockIdx.x;
            int j = threadIdx.x;
            int ind = i * vertexnum + j;
            dist[ind] = 0;
            for (int k = 0; k < 3; k++) {
                float restShape = templateRestShape[j * 3 + k] + shapeBlendShape[j * 3 + k];
                dist[ind] += (curvertices[i * 3 + k] - restShape) * (curvertices[i * 3 + k] - restShape);
            }
        }

        __global__ void FindKNN2(
                const PtrSz<const float>dist,
                const int vertexnum,
                PtrSz<int> ind
        ) {
            int i = threadIdx.x;

            ind[i * 4 + 0] = 0;
            ind[i * 4 + 1] = 1;
            ind[i * 4 + 2] = 2;
            ind[i * 4 + 3] = 3;

            for (int l = 0; l < 4; l++)
                for (int p = 0; p < 3 - l; p++)
                    if (dist[i * vertexnum + ind[i * 4 + p]] > dist[i * vertexnum + ind[i * 4 + p + 1]]) {
                        int tmp = ind[p];
                        ind[p] = ind[p + 1];
                        ind[p + 1] = tmp;
                    }

            //find first 4 minimum distances
            for (int k = 4; k < vertexnum; k++)
                for (int t = 0; t < 4; t++)
                    if (dist[i * vertexnum + k] < dist[i * vertexnum + ind[i * 4 + t]]) {
                        for (int l = 3; l > t; l--)
                            ind[i * 4 + l] = ind[i * 4 + l - 1];
                        ind[i * 4 + t] = k;
                        continue;
                    }
        }

        __global__ void CalculateWeights(
                const PtrSz<const float> dist,
                const PtrSz<const float> weights,
                const PtrSz<const int> ind,
                const int jointnum,
                const int vertexnum,
                PtrSz<float> new_weights
        ) {
            int j = threadIdx.x; // num of weight
            int i = blockIdx.x; // num of vertex

            new_weights[i * jointnum + j] = 0;
            float weight = 0;
            for (int k = 0; k < 4; k++) {
                weight += dist[i * vertexnum + ind[i * 4 + k]];
                new_weights[i * jointnum + j] += dist[i * vertexnum + ind[i * 4 + k]] *
                        weights[ind[i * 4 + k] * jointnum + j];
            }
            new_weights[i * jointnum + j] /= weight;
        }


        __global__ void mark_body_nodes(
                const DeviceArrayView<float4> reference_vertex,
                const PtrSz<const float> templateRestShape,
                const PtrSz<const float> shapeBlendShape,
                const int vertex_num,
                const int max_dist,
                PtrSz<bool> on_body
        ) {
            int i = blockIdx.x; // reference vertex size
            int j = threadIdx.x; // smpl size
            if (3 * (j + 1) >= shapeBlendShape.size || i >= reference_vertex.Size())
                return;

            float dist = 0;
            const float cur[3] = {reference_vertex[i].x, reference_vertex[i].y, reference_vertex[i].z};
            for (int k = 0; k < 3; k++) {
                float restShape = templateRestShape[j * 3 + k] + shapeBlendShape[j * 3 + k];
                dist += (cur[k] - restShape) * (cur[k] - restShape);
            }
            if (dist <= max_dist)
                on_body[i] = true;
        }

        __global__ void copy_body_nodes(
                const DeviceArrayView<float4> reference_vertex,
                const PtrSz<const bool> on_body,
                PtrSz<float4> onbody_points,
                PtrSz<float4> farbody_points
        ) {
            int on = 0, far = 0;
            for (int i = 0; i < reference_vertex.Size(); i++) {
                if (on_body[i])
                    onbody_points[on++] = reference_vertex[i];
                else
                    farbody_points[far++] = reference_vertex[i];
            }
        }
    }

    DeviceArray<float> SMPL::lbs_for_custom_vertices(
            const DeviceArray<float> &beta,
            const DeviceArray<float> &theta,
            const DeviceArray<float> &d_vertices,
            hipStream_t stream
    ) {
        DeviceArray<float> d_shapeBlendShape(DeviceArray<float>(VERTEX_NUM * 3));
        DeviceArray<float> d_dist(DeviceArray<float>(d_vertices.size() * VERTEX_NUM));
        DeviceArray<int> d_ind(DeviceArray<int>(d_vertices.size() * 4));
        DeviceArray<float> d_cur_weights(DeviceArray<float>(d_vertices.size() * JOINT_NUM));
        DeviceArray<float> d_result_vertices(DeviceArray<float>(d_vertices.size() * 3));

        shapeBlendShape(beta, d_shapeBlendShape, stream);
        // find k nearest neigbours
        device::FindKNN1<<<d_vertices.size(),VERTEX_NUM>>>(d_templateRestShape, d_shapeBlendShape, VERTEX_NUM, d_vertices, d_dist);
        device::FindKNN2<<<1,d_vertices.size()>>>(d_dist, VERTEX_NUM, d_ind);
        // calculate weights
        device::CalculateWeights<<<d_vertices.size(),JOINT_NUM>>>(d_dist, d_weights, d_ind,  JOINT_NUM, VERTEX_NUM, d_cur_weights);
        run(beta, theta, d_cur_weights, d_result_vertices, stream, d_vertices);

        return d_result_vertices;
    }

    void SMPL::SplitOnBodyVertices(
            const DeviceArrayView<float4>& reference_vertex,
            const DeviceArray<float> &beta,
            DeviceArray<float4>& onbody_points,
           DeviceArray<float4>& farbody_points,
       	hipStream_t stream
    ) {
	std::cout << "start split\n";

        DeviceArray<float> d_shapeBlendShape = DeviceArray<float>(VERTEX_NUM * 3);
        DeviceArray<bool> marked_vertices = DeviceArray<bool>(reference_vertex.Size());

        shapeBlendShape(beta, d_shapeBlendShape, stream);

        device::mark_body_nodes<<<reference_vertex.Size(),VERTEX_NUM>>>(
                reference_vertex, d_templateRestShape,
                d_shapeBlendShape, VERTEX_NUM, 2.8f * Constants::kNodeRadius, marked_vertices);

        bool *host_array = (bool *)malloc(sizeof(bool) * marked_vertices.size());

	marked_vertices.download(host_array);
        int num = 0;
        for (int i = 0; i < marked_vertices.size(); i++)
            if (host_array[i])
                num++;

        onbody_points = DeviceArray<float4>(num);
        farbody_points = DeviceArray<float4>(reference_vertex.Size() - num);

        device::copy_body_nodes<<<1,1>>>(reference_vertex, marked_vertices, onbody_points, farbody_points);
cudaSafeCall(hipStreamSynchronize(stream));
	std::cout << "end split\n";
    }
}
