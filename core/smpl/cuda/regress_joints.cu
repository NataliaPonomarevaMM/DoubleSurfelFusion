#include "hip/hip_runtime.h"
#include <cmath>
#include "core/smpl/def.h"
#include "core/smpl/smpl.h"

namespace smpl {
    namespace device {
        __global__ void
        RegressJoints1(float *templateRestShape, float *shapeBlendShape, float *poseBlendShape,
                       float *restShape) {
            int j = blockIdx.x;
            int k = threadIdx.x;

            int ind = j * 3 + k;
            restShape[ind] = templateRestShape[ind] + shapeBlendShape[ind] + poseBlendShape[ind];
        }

        __global__ void
        RegressJoints2(float *templateRestShape, float *shapeBlendShape, float *jointRegressor, int vertexnum,
                       float *joints) {
            int j = blockIdx.x;
            int l = threadIdx.x;

            int ind = j * 3 + l;
            joints[ind] = 0;
            for (int k = 0; k < vertexnum; k++)
                joints[ind] += (templateRestShape[k * 3 + l] +
                        shapeBlendShape[k * 3 + l]) * jointRegressor[j * vertexnum + k];
        }
    }

    std::tuple<float *, float *> SMPL::regressJoints(float *d_shapeBlendShape, float *d_poseBlendShape) {
        float *d_joints, *d_restShape;
        hipMalloc((void **) &d_joints, JOINT_NUM * 3 * sizeof(float));
        hipMalloc((void **) &d_restShape, VERTEX_NUM * 3 * sizeof(float));

        device::RegressJoints1<<<VERTEX_NUM,3>>>(d_templateRestShape, d_shapeBlendShape, d_poseBlendShape, d_restShape);
        device::RegressJoints2<<<JOINT_NUM,3>>>(d_templateRestShape, d_shapeBlendShape, d_jointRegressor,
                VERTEX_NUM, d_joints);

        return {d_restShape, d_joints};
    }
}