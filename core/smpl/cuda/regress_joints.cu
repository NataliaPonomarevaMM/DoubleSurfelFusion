#include "hip/hip_runtime.h"
#include <cmath>
#include "core/smpl/def.h"
#include "core/smpl/smpl.h"
#include "common/common_types.h"
#include <>

namespace surfelwarp {
    namespace device {
        __global__ void RegressJoints1(
                const PtrSz<const float> templateRestShape,
                const PtrSz<const float> shapeBlendShape,
                const PtrSz<const float> poseBlendShape,
                PtrSz<float> restShape
        ) {
            const auto ind = threadIdx.x + blockDim.x * blockIdx.x;
            if (ind >= restShape.size)
                return;
            restShape[ind] = templateRestShape[ind] + shapeBlendShape[ind] + poseBlendShape[ind];
        }

        __global__ void RegressJoints2(
                const PtrSz<const float> templateRestShape,
                const PtrSz<const float> shapeBlendShape,
                const PtrSz<const float> jointRegressor,
                const int vertexnum,
                PtrSz<float> joints
        ) {
            const auto ind = threadIdx.x + blockDim.x * blockIdx.x;
            if (ind >= restShape.size)
                return;

            int l = threadIdx.x;
            joints[ind] = 0;
            for (int k = 0; k < vertexnum; k++)
                joints[ind] += (templateRestShape[k * 3 + l] +
                        shapeBlendShape[k * 3 + l]) * jointRegressor[j * vertexnum + k];
        }
    }

    void SMPL::regressJoints(
            const DeviceArray<float> &d_shapeBlendShape,
            const DeviceArray<float> &d_poseBlendShape,
            DeviceArray<float> &d_restShape,
            DeviceArray<float> &d_joints,
            hipStream_t stream
    ) {
        device::RegressJoints1<<<VERTEX_NUM,3,0,stream>>>(d_templateRestShape,
                d_shapeBlendShape, d_poseBlendShape, d_restShape);
        device::RegressJoints2<<<JOINT_NUM,3,0,stream>>>(d_templateRestShape,
                d_shapeBlendShape, d_jointRegressor, VERTEX_NUM, d_joints);
    }
}
