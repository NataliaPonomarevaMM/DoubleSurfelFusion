#include "hip/hip_runtime.h"
#include <cmath>
#include "core/smpl/def.h"
#include "core/smpl/smpl.h"
#include "common/common_types.h"
#include <>

namespace surfelwarp {
    namespace device {
        __global__ void countRS(
                const PtrSz<const float> templateRestShape,
                const PtrSz<const float> shapeBlendShape,
                const PtrSz<const float> poseBlendShape,
                PtrSz<float> restShape
        ) {
            const auto ind = threadIdx.x + blockDim.x * blockIdx.x;
            if (ind >= restShape.size)
                return;
            restShape[ind] = templateRestShape[ind] + shapeBlendShape[ind] + poseBlendShape[ind];
        }

        __global__ void RegressJoints2(
                const PtrSz<const float> templateRestShape,
                const PtrSz<const float> shapeBlendShape,
                const PtrSz<const float> jointRegressor,
                const int vertexnum,
                PtrSz<float> joints
        ) {
            const auto ind = threadIdx.x + blockDim.x * blockIdx.x;
            if (ind >= joints.size)
                return;

		    const auto j = ind / 3;
		    const auto l = ind % 3;
            joints[ind] = 0;
            for (int k = 0; k < vertexnum; k++)
                joints[ind] += (templateRestShape[k * 3 + l] +
                        shapeBlendShape[k * 3 + l]) * jointRegressor[j * vertexnum + k];
        }
    }

    void SMPL::countRestShape(
            const DeviceArray<float> &shapeBlendShape,
            const DeviceArray<float> &poseBlendShape,
            hipStream_t stream
    ) {
        dim3 blk(128);
        dim3 grid(divUp(VERTEX_NUM * 3, blk.x));
        device::countRS<<<grid, blk,0,stream>>>(m__templateRestShape,
                shapeBlendShape, poseBlendShape, m_restShape);
    }

    void SMPL::regressJoints(
            const DeviceArray<float> &shapeBlendShape,
            DeviceArray<float> &joints,
            hipStream_t stream
    ) {
        dim3 blk(128);
        dim3 grid(divUp(VERTEX_NUM * 3, blk.x));
        device::RegressJoints2<<<grid, blk,0,stream>>>(m__templateRestShape,
                shapeBlendShape, m__jointRegressor, VERTEX_NUM, joints);
    }
}
