#include "hip/hip_runtime.h"
#include <cmath>
#include "core/smpl/def.cuh"
#include "core/smpl/smpl.h"
#include "common/common_types.h"
#include <>

namespace surfelwarp {
    namespace device {
        __global__ void RegressJoints1(
                const PtrSz<const float> shapeBlendShape,
                const PtrSz<const float> poseBlendShape,
                PtrSz<float> restShape
        ) {
            const auto ind = threadIdx.x + blockDim.x * blockIdx.x;
            if (ind >= restShape.size)
                return;
            restShape[ind] = m__templateRestShape[ind] + shapeBlendShape[ind] + poseBlendShape[ind];
        }

        __global__ void RegressJoints2(
                const PtrSz<const float> shapeBlendShape,
                const int vertexnum,
                PtrSz<float> joints
        ) {
            const auto ind = threadIdx.x + blockDim.x * blockIdx.x;
            if (ind >= joints.size)
                return;

		 int j = blockIdx.x;
             int l = threadIdx.x;
            joints[ind] = 0;
            for (int k = 0; k < vertexnum; k++)
                joints[ind] += (m__templateRestShape[k * 3 + l] +
                        shapeBlendShape[k * 3 + l]) * m__jointRegressor[j * vertexnum + k];
        }
    }

    void SMPL::regressJoints(
            const DeviceArray<float> &shapeBlendShape,
            const DeviceArray<float> &poseBlendShape,
            DeviceArray<float> &restShape,
            DeviceArray<float> &joints,
            hipStream_t stream
    ) {
        device::RegressJoints1<<<VERTEX_NUM,3,0,stream>>>(shapeBlendShape, poseBlendShape, restShape);
        device::RegressJoints2<<<JOINT_NUM,3,0,stream>>>(shapeBlendShape, VERTEX_NUM, joints);
    }
}
