#include "hip/hip_runtime.h"
#include <cmath>
#include "core/smpl/def.h"
#include "core/smpl/smpl.h"
#include "common/common_types.h"
#include <>

namespace surfelwarp {
    namespace device {
        __global__ void Skinning(
                const PtrSz<const float> restShape,
                const PtrSz<const float> transformation,
                const PtrSz<const float> weights,
                const int jointnum,
                PtrSz<float> vertices
        ) {
            int j = blockIdx.x;

            if (j * 3 + 3 >= vertices.size)
                return;

            float coeffs[16] = {0};
            for (int k = 0; k < 4; k++)
                for (int l = 0; l < 4; l++)
                    for (int t = 0; t < jointnum; t++)
                        coeffs[k * 4 + l] += weights[j * jointnum + t] * transformation[t * 16 + k * 4 + l];

            float homoW = coeffs[15];
            for (int t = 0; t < 3; t++)
                homoW += coeffs[12 + t] * restShape[j * 3 + t];
            for (int k = 0; k < 3; k++) {
                vertices[j * 3 + k] = coeffs[k * 4 + 3];
                for (int t = 0; t < 3; t++)
                    vertices[j * 3 + k] += coeffs[k * 4 + t] * restShape[j * 3 + t];
                vertices[j * 3 + k] /= homoW;
            }
        }
    }

    void SMPL::skinning(
            const DeviceArray<float> &transformation,
            hipStream_t stream
    ) {
        device::Skinning<<<VERTEX_NUM,1,0,stream>>>(m_restShape, transformation,  m__weights,
               JOINT_NUM, m_smpl_vertices);
    }
}
