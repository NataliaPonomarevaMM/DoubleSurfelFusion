#include <cmath>
#include "core/smpl/def.h"
#include "core/smpl/smpl.h"

namespace smpl {
    void SMPL::loadToDevice(float *shapeBlendBasis, float *poseBlendBasis,
            float *templateRestShape, float *jointRegressor,
            int64_t *kinematicTree, float *weights) {
        ///BLEND SHAPE
        hipMalloc((void **) &d_poseBlendBasis, VERTEX_NUM * 3 * POSE_BASIS_DIM * sizeof(float));
        hipMemcpy(d_poseBlendBasis, poseBlendBasis, VERTEX_NUM * 3 * POSE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc((void **) &d_shapeBlendBasis, VERTEX_NUM * 3 * SHAPE_BASIS_DIM * sizeof(float));
        hipMemcpy(d_shapeBlendBasis, shapeBlendBasis, VERTEX_NUM * 3 * SHAPE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);
        ///REGRESS JOINTS
        hipMalloc((void **) &d_templateRestShape, VERTEX_NUM * 3 * sizeof(float));
        hipMalloc((void **) &d_jointRegressor, JOINT_NUM * VERTEX_NUM * sizeof(float));
        hipMemcpy(d_templateRestShape, templateRestShape, VERTEX_NUM * 3  * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_jointRegressor, jointRegressor, JOINT_NUM * VERTEX_NUM * sizeof(float), hipMemcpyHostToDevice);
        ///WORLD TRANSFORMATIONS
        hipMalloc((void **) &d_kinematicTree, 2 * JOINT_NUM * sizeof(int64_t));
        hipMemcpy(d_kinematicTree, kinematicTree, 2 * JOINT_NUM * sizeof(int), hipMemcpyHostToDevice);
        ///SKINNING
        hipMalloc((void **) &d_weights, VERTEX_NUM * JOINT_NUM * sizeof(float));
        hipMemcpy(d_weights, weights, VERTEX_NUM * JOINT_NUM * sizeof(float), hipMemcpyHostToDevice);
    }

    float *SMPL::run(float *beta, float *theta, float *d_custom_weights, float *d_vertices, float vertexnum) {
        auto pbs = poseBlendShape(theta);
        auto d_poseRotation = std::get<0>(pbs);
        auto d_restPoseRotation = std::get<1>(pbs);
        auto d_poseBlendShape = std::get<2>(pbs);

        auto d_shapeBlendShape = shapeBlendShape(beta);

        auto rj = regressJoints(d_shapeBlendShape, d_poseBlendShape);
        auto d_restShape = std::get<0>(rj);
        auto d_joints = std::get<1>(rj);
        hipFree(d_shapeBlendShape);
        hipFree(d_poseBlendShape);
        auto d_transformation = transform(d_poseRotation, d_joints);
        hipFree(d_poseRotation);
        hipFree(d_joints);

        if (d_vertices == nullptr) {
            d_vertices = d_restShape;
            vertexnum = VERTEX_NUM;
        }

        float *res = skinning(d_transformation, d_custom_weights, d_vertices, vertexnum);
        hipFree(d_restShape);
        hipFree(d_transformation);

        return res;
    }

    float *SMPL::lbs_for_model(float *beta, float *theta) {
        return run(beta, theta, d_weights);
    }

    SMPL::~SMPL() {
        if (d_poseBlendBasis != nullptr)
            hipFree(d_poseBlendBasis);
        if (d_shapeBlendBasis != nullptr)
            hipFree(d_shapeBlendBasis);
        if (d_templateRestShape != nullptr)
            hipFree(d_templateRestShape);
        if (d_jointRegressor != nullptr)
            hipFree(d_jointRegressor);
        if (d_weights != nullptr)
            hipFree(d_weights);
        if (d_kinematicTree != nullptr)
            hipFree(d_kinematicTree);
    }
}